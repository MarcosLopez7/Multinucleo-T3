#include "hip/hip_runtime.h"
#include <time.h>
#include "book.h"

void fillMatrix(int *, int);
void printMatrix(int *, int);

__global__ void matrixMult(int *a, int *b, int *c, int *n) {

    int tid_i = threadIdx.x + blockIdx.x * blockDim.x;	
    
    for (int i = 0; i < *n; ++i){ 
	c[(tid_i * (*n)) + i] = 0;
    	for(int j = 0; j < *n; ++j){
	    c[(tid_i * (*n)) + i] += a[(tid_i * (*n)) + j] * b[(j * (*n)) + i];
	}
    }
}

int main(int argc, const char *argv[]) {
 
    if (argc != 3) {
        	printf("Se necesita dos argumentos para correr este programa, el primero para el tamano de las matrices, y "
                "otra que es 1 o 0 para indicar para imprimir o no las matrices\n");
        exit(-1);
    }

    int n = atoi(argv[1]);
    int print = atoi(argv[2]);
    float tiempo1, tiempo2;
    hipEvent_t inicio1, fin1, inicio2, fin2;

    hipEventCreate(&inicio1); 
    hipEventCreate(&fin1);
    hipEventCreate(&inicio2); 
    hipEventCreate(&fin2);
    hipEventRecord( inicio1, 0 );

    int *matA = (int *) malloc(n * n * sizeof(int));
    int *matB = (int *) malloc(n * n * sizeof(int));
    int *matC = (int *) malloc(n * n * sizeof(int));
/*
    for (int i = 0; i < n; ++i) {
        matA[i] = (int *) malloc(n * sizeof(int *));
        matB[i] = (int *) malloc(n * sizeof(int *));
    }
*/
    int *dev_ma, *dev_mb, *dev_mc;// **tempa, **tempb;
    int *dev_n;

//    tempa = (int **) malloc(n * sizeof(int *));
//    tempb = (int **) malloc(n * sizeof(int *));

    HANDLE_ERROR(hipMalloc((void **) &dev_ma, n * n * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **) &dev_mb, n * n * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **) &dev_mc, n * n * sizeof(int)));

    HANDLE_ERROR(hipMalloc((void **) &dev_n, sizeof(int)));

    srand((int) time(NULL));

    fillMatrix(matA, n);
    fillMatrix(matB, n);

    HANDLE_ERROR(hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice));
/*
    for (int i = 0; i < n; ++i) {
	HANDLE_ERROR(hipMalloc((void **) &tempa[i], n * sizeof(int)));
	HANDLE_ERROR(hipMemcpy (tempa[i], matA[i], n * sizeof(int), hipMemcpyHostToDevice));
   	HANDLE_ERROR(hipMalloc((void **) &tempb[i], n * sizeof(int)));
	HANDLE_ERROR(hipMemcpy (tempb[i], matB[i], n * sizeof(int), hipMemcpyHostToDevice));
    }
*/
    HANDLE_ERROR(hipMemcpy (dev_ma, matA, n * n * sizeof(int), hipMemcpyHostToDevice) );
    HANDLE_ERROR(hipMemcpy (dev_mb, matB, n * n * sizeof(int), hipMemcpyHostToDevice) );
    
    hipEventRecord( inicio2, 0 );
    matrixMult<<<n,1>>>(dev_ma, dev_mb, dev_mc, dev_n);
    hipEventRecord( fin2, 0); // Se toma el tiempo final.
    hipEventSynchronize( fin2 ); // Se sincroniza
    hipEventElapsedTime( &tiempo2, inicio2, fin2 );

    HANDLE_ERROR(hipMemcpy(matC, dev_mc, n * n * sizeof(int), hipMemcpyDeviceToHost));

    if (print) {
        printMatrix(matA, n);
        printMatrix(matB, n);
    	printMatrix(matC, n);
    }

    HANDLE_ERROR( hipFree( dev_ma ) );
    HANDLE_ERROR( hipFree( dev_mb ) );
    HANDLE_ERROR( hipFree( dev_mc ) );
    HANDLE_ERROR( hipFree( dev_n ) );
//    free(tempa);
//    free(tempb);
    free(matA);
    free(matB);
    free(matC);

    hipEventRecord( fin1, 0); // Se toma el tiempo final.
    hipEventSynchronize( fin1 ); // Se sincroniza
    hipEventElapsedTime( &tiempo1, inicio1, fin1 );

    printf("Tiempo de cálculo: %f , tiempo total: %f\n", tiempo2, tiempo1);

    return 0;
}

void fillMatrix(int *m, int n) {

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            m[(i * n) + j] = (rand() % 991) + 10;
        }
    }

}

void printMatrix(int *m, int n) {

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("m[%d][%d]= %d ", i, j, m[(i * n) + j]);
        }
        printf("\n");
    }

}

